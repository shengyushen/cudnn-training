#include "hip/hip_runtime.h"
/*
 * This code is released into the public domain.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)



//seems to be a class like struct
class baseModule{
	public : 
	string name;
  int in_channels, out_channels ;
  int in_width, in_height, out_width, out_height;

	float * pin; // this is pass in from outside
	long inbuf_size;
	float * pout; // pout should be alloc by child class
	long outbuf_size;

	int minibatch;

	int gpuid;

  hipdnnHandle_t cudnnHandle;
  hipblasHandle_t cublasHandle;

	size_t m_workspaceSizeByte;
	void * p_workspace;

	bool bNeedSyncInTensor;


	void print() {
					cout<<"name "<<name<<endl;
					cout<<"in_channels "<<in_channels<<endl;
					cout<<"out_channels "<<out_channels<<endl;
					cout<<"in_width "<<in_width<<endl;
					cout<<"in_height "<<in_height<<endl;
					cout<<"out_width "<<out_width<<endl;
					cout<<"out_height "<<out_height<<endl;
					cout<<"inbuf_size "<<inbuf_size<<endl;
					cout<<"outbuf_size "<<outbuf_size<<endl;
					cout<<"minibatch "<<minibatch<<endl;
					cout<<"gpuid "<<gpuid<<endl;
	}
	baseModule(
									string name_,
  								hipdnnHandle_t cudnnHandle_,
								  hipblasHandle_t cublasHandle_,
									int gpuid_,
									int minibatch_,
									int in_c_,
									int out_c_,
									int in_h_,
									int in_w_,
									int out_h_,
									int out_w_,
									float * pin_
									) 
	{
		name=name_;
		cudnnHandle = cudnnHandle_;
		cublasHandle = cublasHandle_;
		gpuid = gpuid_;
		minibatch = minibatch_;
		in_channels = in_c_;
		out_channels = out_c_;
		in_width = in_w_;
		in_height = in_h_;
		out_width = out_w_;
		out_height = out_h_;
		pin = pin_;
		inbuf_size = minibatch_*in_c_*in_w_*in_h_;
		outbuf_size = minibatch_*out_c_*out_w_*out_h_;
		m_workspaceSizeByte=0;
		p_workspace=NULL;
		bNeedSyncInTensor=true;

		assert(gpuid>=0);
		assert(minibatch >0);
		assert(in_channels >0);
		assert(out_channels >0);
		assert(in_width >0);
		assert(in_height >0);
		assert(out_width >0);
		assert(out_height >0);
		assert(pin );

		checkCudaErrors(hipSetDevice(gpuid));
    checkCudaErrors (hipMallocManaged (&pout, sizeof (float) *outbuf_size ));
    checkCudaErrors (hipMemAdvise (pout,sizeof(float)* outbuf_size ,hipMemAdviseSetPreferredLocation,gpuid));
	}

	virtual void run1step() {};

	~baseModule  () {
			checkCudaErrors(hipSetDevice(gpuid));
			hipFree(pout);
	}
	size_t getOutputFloatNumber() {
					return outbuf_size;
	}
	size_t getInputFloatNumber() {
					return inbuf_size;
	}

};

class ConvBiasLayer: public baseModule
{
		public :
		int kernel_size,stride;
		hipdnnTensorDescriptor_t biasTensor;
		float * pconvbias;
		hipdnnTensorDescriptor_t srcTensorDesc;
		hipdnnFilterDescriptor_t filterDesc;
		float * pconvWeigth;
		hipdnnConvolutionDescriptor_t convDesc;
		hipdnnTensorDescriptor_t  dstTensorDesc; //this out already have pout in baseModule
		hipdnnConvolutionFwdAlgo_t algo;

    ConvBiasLayer (
				string name_,
				hipdnnHandle_t cudnnHandle_,
			  hipblasHandle_t cublasHandle_,
				int gpuid_,
				int minibatch_,
				int in_channels_, 
				int in_h_, int in_w_, 
				int numFilter_, int kernel_size_, int stride_, 
				int paddingH_, int paddingW_,
				float * pin_)  // pin pass from outside
						: baseModule(
									name_,
									cudnnHandle_,
									cublasHandle_,
									gpuid_,
									minibatch_,
									in_channels_,
									numFilter_,
									in_h_,
									in_w_,
									(in_h_+paddingH_*2-kernel_size_+1)/stride_,
									(in_w_+paddingW_*2-kernel_size_+1)/stride_,
									pin_
								)
		{
						printf("ConvBiasLayer gpuid %d minibatch %d in_channels_ %d in_h_ %d in_w_ %d numFilter_ %d kernel_size_ %d stride_ %d paddingH_ %d paddingW_ %d\n",
						                      gpuid ,  minibatch ,  in_channels_ ,  in_h_ ,  in_w_ ,  numFilter_ ,  kernel_size_ ,  stride_ ,  paddingH_ ,  paddingW_ );
				assert((in_w_+paddingW_*2-kernel_size_+1)%stride_ == 0);
				assert((in_h_+paddingH_*2-kernel_size_+1)%stride_ == 0);

				kernel_size = kernel_size_;
				assert(kernel_size<16); //this is not strict, just to prevent unreasonable large kernel
				stride=stride_;
				assert(stride < 16);//also not strict

				//bias descriptor
				checkCUDNN (hipdnnCreateTensorDescriptor (&biasTensor));
    		checkCUDNN (hipdnnSetTensor4dDescriptor (biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1));
    		checkCudaErrors(hipMallocManaged(&pconvbias, sizeof(float) * out_channels ));

				//set the source tensor
    		checkCUDNN (hipdnnCreateTensorDescriptor (&srcTensorDesc));
				//this may fail in 800 pixel because the hipdnnSetTensor4dDescriptor require the tensor smaller than 2GB, so I may need 700
    		checkCUDNN (hipdnnSetTensor4dDescriptor (srcTensorDesc,
					    HIPDNN_TENSOR_NCHW,
					    HIPDNN_DATA_FLOAT, minibatch, in_channels, in_height, in_width));

				//set the filter desc
    		checkCUDNN (hipdnnCreateFilterDescriptor (&filterDesc));
    		checkCUDNN (hipdnnSetFilter4dDescriptor (filterDesc,
					    HIPDNN_DATA_FLOAT,
					    HIPDNN_TENSOR_NCHW,
					    out_channels,
					    in_channels,
					    kernel_size,
					    kernel_size));
				checkCudaErrors(hipMallocManaged(&pconvWeigth,sizeof(float)*in_channels_*kernel_size_*kernel_size_*numFilter_));

    		checkCUDNN (hipdnnCreateConvolutionDescriptor (&convDesc));
    		checkCUDNN (hipdnnSetConvolution2dDescriptor (convDesc,
						 paddingH_, paddingW_,
						 stride, stride,
						 1, 1, // we currently dont support dilation
						 HIPDNN_CROSS_CORRELATION,
						 HIPDNN_DATA_FLOAT));

				int n,c,h,w;
    		checkCUDNN (hipdnnGetConvolution2dForwardOutputDim (convDesc,
						       srcTensorDesc,
						       filterDesc,
						       &n, &c, &h, &w));
				assert(n==minibatch);
				assert(c=out_channels);
				assert(h==out_height);
				assert(w==out_width);

    		checkCUDNN (hipdnnCreateTensorDescriptor (&dstTensorDesc));
    		checkCUDNN (hipdnnSetTensor4dDescriptor (dstTensorDesc,
					    HIPDNN_TENSOR_NCHW,
					    HIPDNN_DATA_FLOAT, n, c, h, w));
		    checkCUDNN (hipdnnGetConvolutionForwardAlgorithm (cudnnHandle,
						     srcTensorDesc,
						     filterDesc,
						     convDesc,
						     dstTensorDesc,
						     HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
						     0, &algo));

		    checkCUDNN (hipdnnGetConvolutionForwardWorkspaceSize (cudnnHandle,
							 srcTensorDesc,
							 filterDesc,
							 convDesc,
							 dstTensorDesc,
							 algo, &m_workspaceSizeByte));
		}
	void run1step () {
				assert(p_workspace!=NULL);
				assert(m_workspaceSizeByte!=0);
        float alpha = 1.0f, beta = 0.0f;
        checkCudaErrors(hipSetDevice(gpuid));
				assert(p_workspace);
        checkCUDNN(hipdnnConvolutionForward(cudnnHandle, 
																					&alpha, 
																					srcTensorDesc, pin, 
																					filterDesc, pconvWeigth, 
																					convDesc, 
                                           algo, p_workspace, m_workspaceSizeByte, &beta,
                                           dstTensorDesc, pout));
        checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, biasTensor,
                                  pconvbias, &alpha, dstTensorDesc, pout));

	}
	~ConvBiasLayer() {
		checkCUDNN (hipdnnDestroyTensorDescriptor(biasTensor));
		checkCUDNN (hipdnnDestroyTensorDescriptor(srcTensorDesc));
		checkCUDNN (hipdnnDestroyFilterDescriptor(filterDesc));
		checkCudaErrors(hipFree(pconvWeigth));
		checkCUDNN (hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCUDNN (hipdnnDestroyTensorDescriptor (dstTensorDesc));
	}

};


///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
__global__ void
FillOnes (float *vec, int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;

  vec[idx] = 1.0f;
}

/**
 * Computes the backpropagation results of the Softmax loss for each result in a batch.
 * Uses the softmax values obtained from forward propagation to compute the difference.
 *
 * @param label The training batch label values.
 * @param num_labels The number of possible labels.
 * @param batch_size The size of the trained batch.
 * @param diff The resulting gradient.
 */
__global__ void
SoftmaxLossBackprop (const float *label, int num_labels, int batch_size,
		     float *diff)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= batch_size)
    return;

  const int label_value = static_cast < int >(label[idx]);

  // For each item in the batch, decrease the result of the label's value by 1
  diff[idx * num_labels + label_value] -= 1.0f;
}

///////////////////////////////////////////////////////////////////////////////////////////
// CUDNN/CUBLAS training context

class TrainingContext
{
	public :

  hipdnnHandle_t cudnnHandle;
  hipblasHandle_t cublasHandle;
  int m_gpuid;
  int m_batchSize;
	//only the first tensor is need to store here
  hipdnnTensorDescriptor_t dataTensor;

	std::vector<class baseModule *> vmod;
	int currentlayer;

	void * pworkspace;

	void print() {
					printf("TrainingContext m_gpuid %d m_batchSize %d\n",m_gpuid,m_batchSize);
					for(int i =0;i<vmod.size();i++) {
						vmod[i]->print();
					}
	}

  TrainingContext (int gpuid, int batch_size)
  {
    m_batchSize = batch_size;
		m_gpuid =gpuid;
    printf ("gpuid %d batch_size %d\n", gpuid,batch_size);
		currentlayer=0;

    // Create CUBLAS and CUDNN handles
    checkCudaErrors (hipSetDevice (gpuid));
    checkCudaErrors (hipblasCreate (&cublasHandle));
    checkCUDNN (hipdnnCreate (&cudnnHandle));

    // Create tensor descriptors
    checkCUDNN (hipdnnCreateTensorDescriptor (&dataTensor));
  }

	void addMod(class baseModule * pmod) {
		vmod.push_back(pmod);
	}

	baseModule * getCurrentLayer() {
			assert(currentlayer >=0);
			if(currentlayer >= vmod.size()) {
					cout<<"currentlayer "<<currentlayer<<"vmod.size "<<vmod.size()<<endl;
					assert(0);
			}
			return vmod[currentlayer];
	}

	void reset() {
					currentlayer=0;
	}

	bool isFinished() {
		 if(currentlayer>=vmod.size()) return true;
		 else return false;
	}

	void finishAddMod () {
		size_t maxsize=0;
		for(int i=0;i<vmod.size();i++) {
			maxsize = std::max(maxsize,vmod[i]->m_workspaceSizeByte);
		}
		//alloc new size
		checkCudaErrors(hipMallocManaged(&pworkspace,maxsize));
		for(int i=0;i<vmod.size();i++) {
				vmod[i]->p_workspace = pworkspace;
				vmod[i]->m_workspaceSizeByte=maxsize;
		}
	}

   ~TrainingContext ()
  {
		for(int i=0;i<vmod.size();i++) {
			delete vmod[i];
		}
    checkCudaErrors (hipSetDevice (m_gpuid));

    checkCUDNN (hipdnnDestroyTensorDescriptor (dataTensor));
    checkCudaErrors (hipblasDestroy (cublasHandle));
    checkCUDNN (hipdnnDestroy (cudnnHandle));
  }

	 void ForwardPropagation1() {
		 if(currentlayer>=vmod.size()) {
 		  cout<<"finished at layer "<<currentlayer<<endl;
			assert(0);
		 } else {
				cout<<"layer "<<currentlayer<<endl;
        checkCudaErrors(hipSetDevice(m_gpuid));

        // Conv1 layer
				vmod[currentlayer]->run1step();
				
				currentlayer++;
		 }
	 }
};

__global__ void
ssyinitfloat (float *p, size_t n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (size_t idx = index; idx < n; idx += stride)
    {
      p[idx] = 0.0;
    }
}

///////////////////////////////////////////////////////////////////////////////////////////
// Main function
//#define WIDTH 280
int
main (int argc, char **argv)
{
  if (argc != 7) {
		printf("Usage : cudnnModelParallel.exe <width> <iteration> <minbatch> <channel> <copy or not> <fract to copy>");
		assert(0);
	}
	cout<<"argc "<<argc<<endl;
  size_t width, height;
  width = atoi (argv[1]);
  height = width;
	cout<<"width "<<width<<endl;
  int iterations = atoi (argv[2]);
	int minib = atoi(argv[3]);
	int chnl = atoi(argv[4]);
  bool copy = (atoi (argv[5]) > 0);
  float fract = (atof (argv[6]));

  // Choose GPU
  int num_gpus;
  checkCudaErrors (hipGetDeviceCount (&num_gpus));
	cout<<"num_gpus "<<num_gpus<<endl;

	int deviceId;
//  int numberOfSMs;
	checkCudaErrors(hipSetDevice(0));
	hipGetDevice(&deviceId);
	cout<<"deviceId "<<deviceId<<endl;
//	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	//printf("numberOfSMs %s\n",numberOfSMs);
//  int threadsPerBlock = 256;
//  int numberOfBlocks = 32*80 ;


  std::vector < float *>d_dataV;
	std::vector <TrainingContext * >contextV;

  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
    {
      checkCudaErrors (hipSetDevice (gpuid));
			//alloc the input data
			float * pdata;
			size_t input_sz = minib*chnl*width*height;
			checkCudaErrors(hipMallocManaged(&pdata,sizeof(float)*input_sz));
			//the context for this gpu
      TrainingContext * pcontext = new TrainingContext (gpuid, minib);

      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											"conv1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											chnl,
											height,width, 
											64,1,1,
											0,0,
											pdata
											);
			class ConvBiasLayer * pconv2=new ConvBiasLayer (
											"conv2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pconv1->out_channels,
											pconv1->out_height, pconv1->out_width,
											64,3,1,
											1,1,
											pconv1->pout
											);
			class ConvBiasLayer * pconv3=new ConvBiasLayer (
											"conv3",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pconv2->out_channels,
											pconv2->out_height, pconv2->out_width,
											256,1,1,
											0,0,
											pconv2->pout
											);

			pcontext -> addMod(pconv1);
			pcontext -> addMod(pconv2);
			pcontext -> addMod(pconv3);
			pcontext -> finishAddMod();

      contextV.push_back (pcontext);
			d_dataV.push_back(pdata);
	}

	for (int gpuid = 0; gpuid < num_gpus; gpuid++) {
					contextV[gpuid]-> print();
	}

  checkCudaErrors (hipDeviceSynchronize ());

  // Use SGD to train the network
  auto t1 = std::chrono::high_resolution_clock::now ();
  for (int iter = 0; iter < iterations; ++iter)
  {
		//reset
		for(int gpuid=0;gpuid<num_gpus;gpuid++)     {
						contextV[gpuid]->reset();
		}

		while(true) {
			//run one layer
		  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			{
				assert(contextV[gpuid]->m_gpuid == gpuid);
			  checkCudaErrors (hipSetDevice (gpuid));
			  contextV[gpuid]->ForwardPropagation1 ();
			}
			if(contextV[0]->isFinished()) break;
			
		  if (copy)
			{
			  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			  {
			      //sync n+1 to n
			      checkCudaErrors (hipSetDevice (gpuid));
						baseModule * pcurrent =contextV[gpuid]->getCurrentLayer();
			      size_t sz = sizeof (float) * (pcurrent->getInputFloatNumber() );
						assert(sz>0);
						cout<<"sz "<<sz<<endl;

			      if (gpuid > 0) {
							baseModule * pPrev =contextV[gpuid-1]->getCurrentLayer();
							size_t szPrev = sizeof (float) * (pPrev->getInputFloatNumber() );
							assert(sz==szPrev);
						  checkCudaErrors (hipMemcpyAsync (pcurrent->pin + sz / (2 * sizeof (float)), pPrev->pin, int (fract * sz / 2), hipMemcpyDefault));
						}
			  }
		
			  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			    {
			      checkCudaErrors (hipSetDevice (gpuid));
			      checkCudaErrors (hipDeviceSynchronize ());
			    }
			}
		}
  }				// end of iteration

  checkCudaErrors (hipDeviceSynchronize ());
  auto t2 = std::chrono::high_resolution_clock::now ();

  cout<<"Iteration time: width "<<width
			<<" fract "<<
			(copy?fract:0.0)
			<<" time " << std::chrono::duration_cast < std::chrono::microseconds > (t2 - t1).count () / 1000.0f / iterations
		<<" ms"<<endl;
  return 0;
}
